#include "hip/hip_runtime.h"
// ---------------------------------------------------------
// This code is inspired by Andy Zeng, Princeton University
// https://github.com/andyzeng/tsdf-fusion
// 
// created by Zhenjiang Li  zhenjiang1.li@tum.de
// created at 2025-01-20
// updated at 2025-01-27
// ---------------------------------------------------------
#include <iostream>
#include <fstream>
#include <iomanip>
#include <sstream>
#include <string>
#include "utils.hpp"
#include "VirtualSensor.h"

// CUDA kernel function to integrate a TSDF voxel volume given depth images
__global__
void Integrate(float * cam_K, float * cam2base, float * depth_im,
              int im_height, int im_width, int voxel_grid_dim_x, int voxel_grid_dim_y, int voxel_grid_dim_z,
              float voxel_grid_origin_x, float voxel_grid_origin_y, float voxel_grid_origin_z, float voxel_size, float trunc_margin,
              float * voxel_grid_TSDF, float * voxel_grid_weight) {

  int pt_grid_z = blockIdx.x;
  int pt_grid_y = threadIdx.x;

  for (int pt_grid_x = 0; pt_grid_x < voxel_grid_dim_x; ++pt_grid_x) {

    // Convert voxel center from grid coordinates to base frame camera coordinates
    float pt_base_x = voxel_grid_origin_x + pt_grid_x * voxel_size;
    float pt_base_y = voxel_grid_origin_y + pt_grid_y * voxel_size;
    float pt_base_z = voxel_grid_origin_z + pt_grid_z * voxel_size;

    // Convert from base frame camera coordinates to current frame camera coordinates
    float tmp_pt[3] = {0};
    tmp_pt[0] = pt_base_x - cam2base[0 * 4 + 3];
    tmp_pt[1] = pt_base_y - cam2base[1 * 4 + 3];
    tmp_pt[2] = pt_base_z - cam2base[2 * 4 + 3];
    float pt_cam_x = cam2base[0 * 4 + 0] * tmp_pt[0] + cam2base[1 * 4 + 0] * tmp_pt[1] + cam2base[2 * 4 + 0] * tmp_pt[2];
    float pt_cam_y = cam2base[0 * 4 + 1] * tmp_pt[0] + cam2base[1 * 4 + 1] * tmp_pt[1] + cam2base[2 * 4 + 1] * tmp_pt[2];
    float pt_cam_z = cam2base[0 * 4 + 2] * tmp_pt[0] + cam2base[1 * 4 + 2] * tmp_pt[1] + cam2base[2 * 4 + 2] * tmp_pt[2];

    if (pt_cam_z <= 0)
      continue;

    int pt_pix_x = roundf(cam_K[0 * 3 + 0] * (pt_cam_x / pt_cam_z) + cam_K[0 * 3 + 2]);
    int pt_pix_y = roundf(cam_K[1 * 3 + 1] * (pt_cam_y / pt_cam_z) + cam_K[1 * 3 + 2]);
    if (pt_pix_x < 0 || pt_pix_x >= im_width || pt_pix_y < 0 || pt_pix_y >= im_height)
      continue;

    float depth_val = depth_im[pt_pix_y * im_width + pt_pix_x];

    if (depth_val <= 0 || depth_val > 6)
      continue;

    float diff = depth_val - pt_cam_z;

    if (diff <= -trunc_margin)
      continue;

    // Integrate
    int volume_idx = pt_grid_z * voxel_grid_dim_y * voxel_grid_dim_x + pt_grid_y * voxel_grid_dim_x + pt_grid_x;
    float dist = fmin(1.0f, diff / trunc_margin);
    float weight_old = voxel_grid_weight[volume_idx];
    float weight_new = weight_old + 1.0f;
    voxel_grid_weight[volume_idx] = weight_new;
    voxel_grid_TSDF[volume_idx] = (voxel_grid_TSDF[volume_idx] * weight_old + dist) / weight_new;
  }
}

// Loads a binary file with depth data and generates a TSDF voxel volume (5m x 5m x 5m at 1cm resolution)
// Volume is aligned with respect to the camera coordinates of the first frame (a.k.a. base frame)
int main(int argc, char * argv[]) {
  // Initalize VirtualSensor
  VirtualSensor sensor;
  if (!sensor.Init("../Data/rgbd_dataset_freiburg1_xyz/")) {
      std::cerr << "Failed to initialize VirtualSensor!" << std::endl;
      return -1;
  }

  int im_width = sensor.GetColorImageWidth();  // width of depth image
  int im_height = sensor.GetColorImageHeight(); // height of depth image

  // tsdf_voxel_parameters
  int voxel_grid_dim_x = 1000; // x_dimension of voxel grid
  int voxel_grid_dim_y = 1000; // y_dimension of voxel grid
  int voxel_grid_dim_z = 1000; // z_dimension of voxel grid

  // voxel_origin
  float voxel_grid_origin_x = -2.5f; // voxel grid origin x coordinate
  float voxel_grid_origin_y = -2.5f; // voxel grid origin y coordinate
  float voxel_grid_origin_z =  0.0f;  // voxel grid origin z coordinate

  // voxel_size and truncation distance
  float voxel_size = 0.005f;      // voxel size (edge length)(m)
  float trunc_margin = voxel_size * 5;     // truncation distance(m)

  // get intrinsics of depth camera
  Eigen::Matrix3f depth_intrinsics = sensor.GetDepthIntrinsics();
  std::cout << "depth_intrinsics:" << std::endl;
  std::cout << depth_intrinsics << std::endl;
  float cam_K[3 * 3];
  for (int i = 0; i < 3; ++i) {
      for (int j = 0; j < 3; ++j) {
          cam_K[i * 3 + j] = depth_intrinsics(i, j);
      }
  }

  // get base frame
  Eigen::Matrix4f base2world = sensor.GetBaseFrame().inverse();
  std::cout << "base2world:" << std::endl;
  std::cout << base2world << std::endl;
  float base2world_mat[16];
  for (int i = 0; i < 4; ++i) {
      for (int j = 0; j < 4; ++j) {
          base2world_mat[i * 4 + j] = base2world(i, j);
      }
  }

  // camputer inverse of base2world
  float base2world_inv[16];
  invert_matrix(base2world_mat, base2world_inv);

  // camputer inverse of base2world
  float * voxel_grid_TSDF = new float[voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z];
  float * voxel_grid_weight = new float[voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z];
  for (int i = 0; i < voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z; ++i)
      voxel_grid_TSDF[i] = 1.0f;
  memset(voxel_grid_weight, 0, sizeof(float) * voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z);

  // camputer inverse of base2world
  float * gpu_voxel_grid_TSDF;
  float * gpu_voxel_grid_weight;
  hipMalloc(&gpu_voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float));
  hipMalloc(&gpu_voxel_grid_weight, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float));
  checkCUDA(__LINE__, hipGetLastError());
  hipMemcpy(gpu_voxel_grid_TSDF, voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gpu_voxel_grid_weight, voxel_grid_weight, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyHostToDevice);
  checkCUDA(__LINE__, hipGetLastError());
  float * gpu_cam_K;
  float * gpu_cam2base;
  float * gpu_depth_im;
  hipMalloc(&gpu_cam_K, 3 * 3 * sizeof(float));
  hipMemcpy(gpu_cam_K, cam_K, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc(&gpu_cam2base, 4 * 4 * sizeof(float));
  hipMalloc(&gpu_depth_im, im_height * im_width * sizeof(float));
  checkCUDA(__LINE__, hipGetLastError());

  // camputer inverse of base2world
  int frame_cnt = 0;
  while (sensor.ProcessNextFrame()) {
    frame_cnt++;
    // Get depth image and camera pose
    float* depth_im = sensor.GetDepth();    

    // Get camera pose
    Eigen::Matrix4f cam2world = sensor.GetTrajectory().inverse();
    float cam2world_mat[16];
    for (int i = 0; i < 4; ++i) {
        for (int j = 0; j < 4; ++j) {
            cam2world_mat[i * 4 + j] = cam2world(i, j);
        }
    }

    // Compute transformation matrices
    float cam2base[16];
    multiply_matrix(base2world_inv,cam2world_mat,  cam2base);
    // print_matrix(cam2base, 4, 4);
    std::cout << "cam2base (4x4):" << std::endl;
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            std::cout << cam2base[i * 4 + j] << " ";
        }
        std::cout << std::endl;
    }

    // Eigen::Matrix4f cam2base_compare = sensor.GetCam2Base();
    // std::cout << "cam2base_compare:" << std::endl;
    // std::cout << cam2base_compare << std::endl;

    // // Inverse of cam2base
    // float cam2base_inv[16] = {0};
    // invert_matrix(cam2base, cam2base_inv);

    // Copy data to GPU
    // hipMemcpy(gpu_cam2base, cam2base_inv, 4 * 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_cam2base, cam2base, 4 * 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(gpu_depth_im, depth_im, im_height * im_width * sizeof(float), hipMemcpyHostToDevice);
    checkCUDA(__LINE__, hipGetLastError());


    std::cout << "Fusing: " << std::endl;
    std::cout << "voxel_grid_dim_z: " << voxel_grid_dim_z << ", voxel_grid_dim_y: " << voxel_grid_dim_y << std::endl;

    // Launch CUDA kernel
    Integrate <<< voxel_grid_dim_z, voxel_grid_dim_y >>>(gpu_cam_K, gpu_cam2base, gpu_depth_im,
                                                        im_height, im_width, voxel_grid_dim_x, voxel_grid_dim_y, voxel_grid_dim_z,
                                                        voxel_grid_origin_x, voxel_grid_origin_y, voxel_grid_origin_z, voxel_size, trunc_margin,
                                                        gpu_voxel_grid_TSDF, gpu_voxel_grid_weight);
    // if(frame_cnt == 10) {
    //   break;      
    // }                              
  }
    // Load TSDF voxel grid from GPU to CPU memory
  hipMemcpy(voxel_grid_TSDF, gpu_voxel_grid_TSDF, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(voxel_grid_weight, gpu_voxel_grid_weight, voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z * sizeof(float), hipMemcpyDeviceToHost);
  checkCUDA(__LINE__, hipGetLastError());

  // save TSDF voxel grid to disk as binary file
  SaveVoxelGrid2SurfacePointCloud("tsdf.ply", voxel_grid_dim_x, voxel_grid_dim_y, voxel_grid_dim_z,
                                  voxel_size, voxel_grid_origin_x, voxel_grid_origin_y, voxel_grid_origin_z,
                                  voxel_grid_TSDF, voxel_grid_weight, 0.2f, 0.0f);
  // Compute surface points from TSDF voxel grid and save to point cloud .ply file(ASCII)
  std::cout << "Saving surface point cloud (tsdf_ASCII.ply)..." << std::endl;
  SaveVoxelGrid2SurfacePointCloud_ASCII("tsdf_ASCII.ply", voxel_grid_dim_x, voxel_grid_dim_y, voxel_grid_dim_z, 
                                  voxel_size, voxel_grid_origin_x, voxel_grid_origin_y, voxel_grid_origin_z,
                                  voxel_grid_TSDF, voxel_grid_weight, 0.2f, 0.0f);


  // Save TSDF voxel grid and its parameters to disk as binary file (float array)
  std::cout << "Saving TSDF voxel grid values to disk (tsdf.bin)..." << std::endl;
  std::string voxel_grid_saveto_path = "tsdf.bin";
  std::ofstream outFile(voxel_grid_saveto_path, std::ios::binary | std::ios::out);
  float voxel_grid_dim_xf = (float) voxel_grid_dim_x;
  float voxel_grid_dim_yf = (float) voxel_grid_dim_y;
  float voxel_grid_dim_zf = (float) voxel_grid_dim_z;
  outFile.write((char*)&voxel_grid_dim_xf, sizeof(float));
  outFile.write((char*)&voxel_grid_dim_yf, sizeof(float));
  outFile.write((char*)&voxel_grid_dim_zf, sizeof(float));
  outFile.write((char*)&voxel_grid_origin_x, sizeof(float));
  outFile.write((char*)&voxel_grid_origin_y, sizeof(float));
  outFile.write((char*)&voxel_grid_origin_z, sizeof(float));
  outFile.write((char*)&voxel_size, sizeof(float));
  outFile.write((char*)&trunc_margin, sizeof(float));
  for (int i = 0; i < voxel_grid_dim_x * voxel_grid_dim_y * voxel_grid_dim_z; ++i)
    outFile.write((char*)&voxel_grid_TSDF[i], sizeof(float));
  outFile.close();

    return 0;
}
